#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
// #define DEBUG
#define UINT uint32_t
#define MAXN 1024
#define Size 1024
#define b    8
 
 
 
int N;
__global__ void matrixmul(UINT A[], UINT B[], UINT C[], int N)
{
    int row = blockIdx.x * b + threadIdx.x;
    int column = blockIdx.y * b + threadIdx.y;
    __shared__ int sA[b][b];
    __shared__ int sB[b][b];
 
    int sum = 0;
    for (int r = 0; r < N / b; r++) {
        sA[threadIdx.x][threadIdx.y] = A[row * N + r * b + threadIdx.y];
        sB[threadIdx.x][threadIdx.y] = B[(r * b  + threadIdx.x) * N + column];
        __syncthreads();
        for (int k = 0; k < b; k++)
            sum += sA[threadIdx.x][k] * sB[k][threadIdx.y];
        __syncthreads();
  }
  C[row * N + column] = sum;
 
}
 
void rand_gen(UINT c, int N, UINT A[]) {
    UINT x = 2, n = N*N;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            x = (x * x + c + i + j)%n;
            A[i * N + j] = x;
        }
    }
}
void print_matrix(int N, UINT A[]) {
    for (int i = 0; i < N; i++) {
        fprintf(stderr, "[");
        for (int j = 0; j < N; j++)
            fprintf(stderr, " %u", A[i * N + j]);
        fprintf(stderr, " ]\n");
    }
}
UINT signature(int N, UINT A[]) {
    UINT h = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            h = (h + A[i * N + j]) * 2654435761LU;
    }
    return h;
}
UINT A[MAXN * MAXN], B[MAXN * MAXN], C[MAXN * MAXN];
int main() {
    uint32_t S1, S2;
    scanf("%d %u %u", &N, &S1, &S2);
    rand_gen(S1, N, A);
    rand_gen(S2, N, B);
 
    int size = N * Size * sizeof(UINT);
    UINT *cuA, *cuB, *cuC;
    hipMalloc((void**)&cuA, size);
    hipMalloc((void**)&cuB, size);
    hipMalloc((void**)&cuC, size);
 
    hipMemcpy(cuA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(cuB, B, size, hipMemcpyHostToDevice);
    dim3 blocks(b, b);
    dim3 grid(N / b, N / b);
    matrixmul <<< grid, blocks >>> (cuA, cuB, cuC, N);
 
    hipMemcpy(C, cuC, size, hipMemcpyDeviceToHost);
 
    //print_matrix(N, C);
    printf("%u\n", signature(N, C));
    return 0;
}