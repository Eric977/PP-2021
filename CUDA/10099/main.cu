#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <omp.h>
#include <inttypes.h>
#include <stdint.h>
#include "utils.h"
 
#define MAXGPU 8
#define MAXCODESZ 32767
#define MAXN 16777216
#define LOCALSIZE 32
uint32_t A[MAXN], B[MAXN], C[MAXN];
 
 
__device__ static inline uint32_t Rotate_left(uint32_t x, uint32_t n) {
    return  (x << n) | (x >> (32-n));
}
__device__ static inline uint32_t Encrypt(uint32_t m, uint32_t key) {
    return (Rotate_left(m, key&31) + key)^key;
}
 
__global__ void vecdot(int key1, int key2, int N, uint32_t* C){
        __shared__ int partial_sum[LOCALSIZE];
        int localID = threadIdx.x;
        int gid = blockIdx.x * blockDim.x + threadIdx.x;
 
 
        int l = gid * LOCALSIZE, r = l + LOCALSIZE;
        int sum = 0;
        for (int i = l; i < r && i < N; ++ i){
                sum += Encrypt(i, key1) * Encrypt(i, key2);
        }
        partial_sum[localID] = sum;
        __syncthreads();
 
        #pragma unroll 8
        for (int i = LOCALSIZE >> 1; i > 0; i >>= 1){
                if (localID < i){
                        partial_sum[localID] += partial_sum[localID + i];
                }
                __syncthreads();
        }
        if (localID == 0){
                C[blockIdx.x] = partial_sum[0];
        }
 
}
int main(int argc, char *argv[]) {
    omp_set_num_threads(4);
    int N;
    uint32_t key1, key2;
        uint32_t *cuC;           
    hipMalloc((void**)&cuC, MAXN / LOCALSIZE);
 
        while (scanf("%d %" PRIu32 " %" PRIu32, &N, &key1, &key2) == 3) {    
                int n = (N - 1) / LOCALSIZE + 1; 
 
                dim3 block(LOCALSIZE);
                dim3 grid(n);
                vecdot<<<grid, block>>> (key1, key2, N, cuC);
                hipMemcpy(C, cuC, n, hipMemcpyDeviceToHost);
                uint32_t sum = 0;
                #pragma omp parallel for schedule(static) reduction(+: sum)
                for (int i = 0; i < n; i ++){
                        sum += C[i];
                }
                printf("%" PRIu32 "\n", sum);
    }
    return 0;
}