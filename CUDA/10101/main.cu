#include <stdio.h>
#include <hip/hip_runtime.h>
#define MAXN 2048
#define LOCALSIZE 16
#define Size 16
int n, N, M;
__global__ void simulate(char A[MAXN][MAXN], char B[MAXN][MAXN], int N, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int localI = threadIdx.x;
    int localJ = threadIdx.y;
    __shared__ char buf[LOCALSIZE][LOCALSIZE];
    int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
    int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};
    char cur = A[i][j];
    buf[localI][localJ] = cur;
    __syncthreads();
        char count = 0;
    for (int k = 0; k < 8; k ++){
        int x = localI + dx[k];
        int y = localJ + dy[k];
        int origin_x = i + dx[k];
        int origin_y = j + dy[k];
 
        if (origin_x < 0 || origin_x >= N || origin_y < 0 || origin_y >= N)
            continue;
        if (x >= 0 && x < Size && y >= 0 && y < Size)
            count += buf[x][y];
        else
            count += A[origin_x][origin_y];
 
    }
    B[i][j] = (count == 3) || (cur && count == 2);
 
}
 
char A[2][MAXN][MAXN];
 
void print(char A[][MAXN][MAXN], int cur, int n){
    for (int i = 0; i < N; i ++){
        for (int j = 0; j < N; j ++){
            printf("%c", A[cur][i][j] + '0');
        }
        printf("\n");
    }
    return;
 
}
 
int main() {
    scanf("%d%d", &N, &M);
 
        int size = MAXN * MAXN * sizeof(char);
 
    n = (1 + N / LOCALSIZE) * LOCALSIZE;
    for (int i = 0; i < N; i ++){
        scanf("%s", A[0][i]);
        for (int j = 0; j < N; j ++){
                A[0][i][j] -= '0';
                }
    }
        char *cuA[2];
        hipMalloc((void**)&cuA[0], size);
        hipMalloc((void**)&cuA[1], size);
        hipMemcpy(cuA[0], A[0], size, hipMemcpyHostToDevice);
        hipMemcpy(cuA[1], A[1], size, hipMemcpyHostToDevice);
 
        int local = LOCALSIZE;
        dim3 block(local, local);
        dim3 grid(n / local, n / local);
        int flag = 0;
        for (int i = 0; i < M; i ++){
                simulate<<<grid, block>>>((char (*)[MAXN])cuA[flag], (char (*)[MAXN])cuA[!flag], N, n);
                flag = !flag;
        }
    hipDeviceSynchronize();
 
        hipMemcpy(A[flag], cuA[flag], size, hipMemcpyDeviceToHost);
        print(A, flag, n);
        return 0;
}