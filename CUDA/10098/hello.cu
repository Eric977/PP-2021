#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 
 
int main(){
    int nDevices;
    hipGetDeviceCount(&nDevices);
    printf("%d devices found supporting CUDA\n", nDevices);
 
    hipDeviceProp_t devInfo;
    for (int i = 0; i < nDevices; i ++){
      hipDeviceProp_t devInfo;
      hipGetDeviceProperties(&devInfo, i);
      printf("----------------------------------\n");
      printf("Device %s\n", devInfo.name);
      printf("----------------------------------\n");
      printf(" Device memory: \t%zu\n", devInfo.totalGlobalMem);
      printf(" Memory per-block: \t%zu\n", devInfo.sharedMemPerBlock);
      printf(" Register per-block: \t%d\n", devInfo.regsPerBlock);
      printf(" Warp size: \t\t%d\n", devInfo.warpSize);
      printf(" Memory pitch: \t\t%zu\n", devInfo.memPitch);
      printf(" Constant Memory: \t%zu\n", devInfo.totalConstMem);
      printf(" Max thread per-block: \t%d\n", devInfo.maxThreadsPerBlock);
      printf(" Max thread dim: \t%d / %d / %d\n", 
          devInfo.maxThreadsDim[0], devInfo.maxThreadsDim[1], devInfo.maxThreadsDim[2]);
      printf(" Max grid size: \t%d / %d / %d\n", 
          devInfo.maxGridSize[0], devInfo.maxGridSize[1], devInfo.maxGridSize[2]);
      printf(" Ver: \t\t\t%d.%d\n", devInfo.major, devInfo.minor);
      printf(" Clock: \t\t%d\n", devInfo.clockRate);
      printf(" Texture Alignment: \t%zu\n", devInfo.textureAlignment);
 
 
    }
 
}